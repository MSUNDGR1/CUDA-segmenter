#include "hip/hip_runtime.h"
#include "segmenter.cuh"
#define BLOCK_SIZE 16
#include <stdlib.h>
/*
Images will be stored in row-major (row*rowWidth + column = index) form
*/


__global__ void meanIMG(uint8_t* img, int rows, int cols, float* rowMeans) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < rows) {
		float sum = 0.0;
		for (int i = 0; i < cols; i++) {
			sum += float(img[index * cols + i]);
			if (index == 0) {
				//printf("sum:  %f, i: %d\n", sum, i);
			}
		}
		rowMeans[index] = sum / cols;
		if (index == 0) {
			//printf("rowMean: %f", rowMeans[index]);
		}
	}
}

__global__ void varIMG(uint8_t* img, int rows, int cols, float mean, float* varsums) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < rows) {
		float sum = 0.0;
		for (int i = 0; i < cols; i++) {
			sum += pow(float(img[index * cols + i]) - mean, 2);
		}
		varsums[index] = sum / cols;
	}
}


__global__ void laplace(unsigned char* img, unsigned char* imgOut, int rows, int cols) {

	int rowInd = blockIdx.x * blockDim.x + threadIdx.x;
	int colInd = blockIdx.y * blockDim.y + threadIdx.y;
	int realInd = rowInd * cols + colInd;

	if (rowInd < rows && colInd < cols) {
		int maxInd = rows * cols - 1;
		int sum = 0;
		sum += 4 * img[realInd];
		if (rowInd > 0) {
			realInd = (rowInd - 1) * cols + colInd;

			sum += -1 * img[realInd];
		}
		if (rowInd < rows - 1) {
			realInd = (rowInd + 1) * cols + colInd;
			sum += -1 * img[realInd];
		}
		if (colInd > 0) {
			realInd = rowInd * cols + (colInd - 1);

			sum += -1 * img[realInd];
		}
		if (colInd < cols - 1) {
			realInd = rowInd * cols + (colInd + 1);

			sum += -1 * img[realInd];
		}
		realInd = rowInd * cols + colInd;

		if (sum < 255) {
			imgOut[realInd] = (unsigned char)sum;
		}
		else {
			imgOut[realInd] = 255;
		}
		//printf("realInd: %d, col: %d\n", realInd, colInd);
	}
}



__global__ void ngbrCompLEA(unsigned char* img, int* clusters, int* clusterSize, int rows, int cols, float diff, int* newClust, float* clusterMeans) {

	int rowInd = blockIdx.x * blockDim.x + threadIdx.x;
	int colInd = blockIdx.y * blockDim.y + threadIdx.y;
	int realInd = rowInd * cols + colInd;

	if (rowInd < rows && colInd < cols && colInd != 0 && colInd % 2 == 0) {
		int nextInd = rowInd * cols + (colInd - 1);
		float currClustMean = clusterMeans[clusters[realInd]];
		if ((clusters[nextInd] = nextInd && abs(currClustMean - int(img[nextInd])) < diff)
			|| abs(currClustMean - int(img[nextInd])) < abs(clusterMeans[clusters[nextInd]] - int(img[nextInd]))){
		
			newClust[nextInd] = clusters[realInd];
			
		}
	}
}

__global__ void ngbrCompLOA(unsigned char* img, int* clusters, int* clusterSize, int rows, int cols, float diff, int* newClust, float* clusterMeans) {

	int rowInd = blockIdx.x * blockDim.x + threadIdx.x;
	int colInd = blockIdx.y * blockDim.y + threadIdx.y;
	int realInd = rowInd * cols + colInd;

	if (rowInd < rows && colInd < cols && colInd != 0 && colInd % 2 == 1) {
		int nextInd = rowInd * cols + (colInd - 1);
		float currClustMean = clusterMeans[clusters[realInd]];
		if ((clusters[nextInd] = nextInd && abs(currClustMean - int(img[nextInd])) < diff)
			|| abs(currClustMean - int(img[nextInd])) < abs(clusterMeans[clusters[nextInd]] - int(img[nextInd]))) {

			newClust[nextInd] = clusters[realInd];

		}
	}
}

__global__ void ngbrCompREA(unsigned char* img, int* clusters, int* clusterSize, int rows, int cols, float diff, int* newClust, float* clusterMeans) {

	int rowInd = blockIdx.x * blockDim.x + threadIdx.x;
	int colInd = blockIdx.y * blockDim.y + threadIdx.y;
	int realInd = rowInd * cols + colInd;

	if (rowInd < rows && colInd < cols - 1 && colInd % 2 == 0) {
		int nextInd = rowInd * cols + (colInd + 1);
		float currClustMean = clusterMeans[clusters[realInd]];
		if ((clusters[nextInd] = nextInd && abs(currClustMean - int(img[nextInd])) < diff)
			|| abs(currClustMean - int(img[nextInd])) < abs(clusterMeans[clusters[nextInd]] - int(img[nextInd]))) {

			newClust[nextInd] = clusters[realInd];

		}
	}
}

__global__ void ngbrCompROA(unsigned char* img, int* clusters, int* clusterSize, int rows, int cols, float diff, int* newClust, float* clusterMeans) {

	int rowInd = blockIdx.x * blockDim.x + threadIdx.x;
	int colInd = blockIdx.y * blockDim.y + threadIdx.y;
	int realInd = rowInd * cols + colInd;

	if (rowInd < rows && colInd < cols - 1 && colInd % 2 == 1) {
		int nextInd = rowInd * cols + (colInd + 1);
		float currClustMean = clusterMeans[clusters[realInd]];
		if ((clusters[nextInd] = nextInd && abs(currClustMean - int(img[nextInd])) < diff)
			|| abs(currClustMean - int(img[nextInd])) < abs(clusterMeans[clusters[nextInd]] - int(img[nextInd]))) {

			newClust[nextInd] = clusters[realInd];

		}
	}
}

__global__ void ngbrCompUEA(unsigned char* img, int* clusters, int* clusterSize, int rows, int cols, float diff, int* newClust, float* clusterMeans) {

	int rowInd = blockIdx.x * blockDim.x + threadIdx.x;
	int colInd = blockIdx.y * blockDim.y + threadIdx.y;
	int realInd = rowInd * cols + colInd;

	if (rowInd < rows && colInd < cols && rowInd != 0 && rowInd % 2 == 0) {
		int nextInd = (rowInd - 1) * cols + (colInd);
		float currClustMean = clusterMeans[clusters[realInd]];
		if ((clusters[nextInd] = nextInd && abs(currClustMean - int(img[nextInd])) < diff)
			|| abs(currClustMean - int(img[nextInd])) < abs(clusterMeans[clusters[nextInd]] - int(img[nextInd]))) {

			newClust[nextInd] = clusters[realInd];

		}
	}
}

__global__ void ngbrCompUOA(unsigned char* img, int* clusters, int* clusterSize, int rows, int cols, float diff, int* newClust, float* clusterMeans) {

	int rowInd = blockIdx.x * blockDim.x + threadIdx.x;
	int colInd = blockIdx.y * blockDim.y + threadIdx.y;
	int realInd = rowInd * cols + colInd;

	if (rowInd < rows && colInd < cols && rowInd != 0 && rowInd % 2 == 1) {
		int nextInd = (rowInd - 1) * cols + (colInd);
		float currClustMean = clusterMeans[clusters[realInd]];
		if ((clusters[nextInd] = nextInd && abs(currClustMean - int(img[nextInd])) < diff)
			|| abs(currClustMean - int(img[nextInd])) < abs(clusterMeans[clusters[nextInd]] - int(img[nextInd]))) {

			newClust[nextInd] = clusters[realInd];

		}
	}
}

__global__ void ngbrCompDEA(unsigned char* img, int* clusters, int* clusterSize, int rows, int cols, float diff, int* newClust, float* clusterMeans) {

	int rowInd = blockIdx.x * blockDim.x + threadIdx.x;
	int colInd = blockIdx.y * blockDim.y + threadIdx.y;
	int realInd = rowInd * cols + colInd;

	if (rowInd < rows-1 && colInd < cols && rowInd != 0 && rowInd % 2 == 0) {
		int nextInd = (rowInd + 1) * cols + (colInd);
		float currClustMean = clusterMeans[clusters[realInd]];
		if ((clusters[nextInd] = nextInd && abs(currClustMean - int(img[nextInd])) < diff)
			|| abs(currClustMean - int(img[nextInd])) < abs(clusterMeans[clusters[nextInd]] - int(img[nextInd]))) {

			newClust[nextInd] = clusters[realInd];

		}
	}
}

__global__ void ngbrCompDOA(unsigned char* img, int* clusters, int* clusterSize, int rows, int cols, float diff, int* newClust, float* clusterMeans) {

	int rowInd = blockIdx.x * blockDim.x + threadIdx.x;
	int colInd = blockIdx.y * blockDim.y + threadIdx.y;
	int realInd = rowInd * cols + colInd;

	if (rowInd < rows - 1 && colInd < cols && rowInd != 0 && rowInd % 2 == 1) {
		int nextInd = (rowInd + 1) * cols + (colInd);
		float currClustMean = clusterMeans[clusters[realInd]];
		if ((clusters[nextInd] = nextInd && abs(currClustMean - int(img[nextInd])) < diff)
			|| abs(currClustMean - int(img[nextInd])) < abs(clusterMeans[clusters[nextInd]] - int(img[nextInd]))) {

			newClust[nextInd] = clusters[realInd];

		}
	}
}


__global__ void ngbrCompLUE(unsigned char* img, int* clusters, int* clusterSize, int rows, int cols, float diff, int* newClust, float* clusterMeans) {

	int rowInd = blockIdx.x * blockDim.x + threadIdx.x;
	int colInd = blockIdx.y * blockDim.y + threadIdx.y;
	int realInd = rowInd * cols + colInd;

	if (rowInd < rows && colInd < cols && colInd != 0 && colInd % 2 == 0 && rowInd != 0 && rowInd %2==0) {
		int nextInd = (rowInd-1) * cols + (colInd - 1);
		float currClustMean = clusterMeans[clusters[realInd]];
		if (abs(currClustMean - int(img[nextInd])) < diff && clusters[realInd] != clusters[nextInd]) {
			if (clusters[nextInd] = nextInd || abs(currClustMean - int(img[nextInd])) < abs(clusterMeans[clusters[nextInd]] - int(img[nextInd]))) {
				newClust[nextInd] = clusters[realInd];
			}
		}
	}
}

__global__ void ngbrCompRUE(unsigned char* img, int* clusters, int* clusterSize, int rows, int cols, float diff, int* newClust, float* clusterMeans) {

	int rowInd = blockIdx.x * blockDim.x + threadIdx.x;
	int colInd = blockIdx.y * blockDim.y + threadIdx.y;
	int realInd = rowInd * cols + colInd;

	if (rowInd < rows && colInd < cols-1  && colInd % 2 == 0 && rowInd != 0 && rowInd % 2 == 0) {
		int nextInd = (rowInd -1) * cols + (colInd + 1);
		float currClustMean = clusterMeans[clusters[realInd]];
		if (abs(currClustMean - int(img[nextInd])) < diff && clusters[realInd] != clusters[nextInd]) {
			if (clusters[nextInd] = nextInd || abs(currClustMean - int(img[nextInd])) < abs(clusterMeans[clusters[nextInd]] - int(img[nextInd]))) {
				newClust[nextInd] = clusters[realInd];
			}
		}
	}
}

__global__ void ngbrCompLDE(unsigned char* img, int* clusters, int* clusterSize, int rows, int cols, float diff, int* newClust, float* clusterMeans) {

	int rowInd = blockIdx.x * blockDim.x + threadIdx.x;
	int colInd = blockIdx.y * blockDim.y + threadIdx.y;
	int realInd = rowInd * cols + colInd;

	if (rowInd < rows-1 && colInd < cols && colInd != 0 && colInd % 2 == 0 && rowInd % 2 == 0) {
		int nextInd = (rowInd + 1) * cols + (colInd - 1);
		float currClustMean = clusterMeans[clusters[realInd]];
		if (abs(currClustMean - int(img[nextInd])) < diff && clusters[realInd] != clusters[nextInd]) {
			if (clusters[nextInd] = nextInd || abs(currClustMean - int(img[nextInd])) < abs(clusterMeans[clusters[nextInd]] - int(img[nextInd]))) {
				newClust[nextInd] = clusters[realInd];
			}
		}
	}
}

__global__ void ngbrCompRDE(unsigned char* img, int* clusters, int* clusterSize, int rows, int cols, float diff, int* newClust, float* clusterMeans) {

	int rowInd = blockIdx.x * blockDim.x + threadIdx.x;
	int colInd = blockIdx.y * blockDim.y + threadIdx.y;
	int realInd = rowInd * cols + colInd;

	if (rowInd < rows-1 && colInd < cols-1 && colInd % 2 == 0 && rowInd % 2 == 0) {
		int nextInd = (rowInd + 1) * cols + (colInd + 1);
		float currClustMean = clusterMeans[clusters[realInd]];
		if (abs(currClustMean - int(img[nextInd])) < diff && clusters[realInd] != clusters[nextInd]) {
			if (clusters[nextInd] = nextInd || abs(currClustMean - int(img[nextInd])) < abs(clusterMeans[clusters[nextInd]] - int(img[nextInd]))) {
				newClust[nextInd] = clusters[realInd];
			}
		}
	}
}


__global__ void ngbrCompLE(unsigned char* img, int* clusters, int* clusterSize, int rows, int cols, float diff, int* newClust, float* clusterMeans) {

	int rowInd = blockIdx.x * blockDim.x + threadIdx.x;
	int colInd = blockIdx.y * blockDim.y + threadIdx.y;
	int realInd = rowInd * cols + colInd;

	if (rowInd < rows && colInd < cols && colInd !=0 && colInd %2 ==0) {
		int nextInd = rowInd * cols + (colInd - 1);
		float currClustMean = clusterMeans[clusters[realInd]];
		if (abs(currClustMean - int(img[nextInd])) < diff && clusters[realInd] != clusters[nextInd]) {
			if (clusters[nextInd] = nextInd || abs(currClustMean - int(img[nextInd])) < abs(clusterMeans[clusters[nextInd]] - int(img[nextInd]))) {
				newClust[nextInd] = clusters[realInd];
			}
		}
	}
}

__global__ void ngbrCompLO(unsigned char* img, int* clusters, int* clusterSize, int rows, int cols, float diff, int* newClust, float* clusterMeans) {

	int rowInd = blockIdx.x * blockDim.x + threadIdx.x;
	int colInd = blockIdx.y * blockDim.y + threadIdx.y;
	int realInd = rowInd * cols + colInd;

	if (rowInd < rows && colInd < cols && colInd != 0 && colInd % 2 == 1) {
		int nextInd = rowInd * cols + (colInd - 1);
		float currClustMean = clusterMeans[clusters[realInd]];
		if (abs(currClustMean - int(img[nextInd])) < diff && clusters[realInd] != clusters[nextInd]) {
			if (clusters[nextInd] = nextInd || abs(currClustMean - int(img[nextInd])) < abs(clusterMeans[clusters[nextInd]] - int(img[nextInd]))) {
				newClust[nextInd] = clusters[realInd];
			}
		}
	}
}

__global__ void ngbrCompRE(unsigned char* img, int* clusters, int* clusterSize,  int rows, int cols, float diff, int* newClust, float* clusterMeans) {

	int rowInd = blockIdx.x * blockDim.x + threadIdx.x;
	int colInd = blockIdx.y * blockDim.y + threadIdx.y;
	int realInd = rowInd * cols + colInd;



	if (rowInd < rows && colInd < cols -1 && colInd % 2 == 0) {
		int nextInd = rowInd * cols + (colInd + 1);
		
		float currClustMean = clusterMeans[clusters[realInd]];
		
		if (abs(currClustMean - int(img[nextInd])) < diff && clusters[realInd] != clusters[nextInd]) {
			if (clusters[nextInd] = nextInd || abs(currClustMean - int(img[nextInd])) < abs(clusterMeans[clusters[nextInd]] - int(img[nextInd]))) {
				newClust[nextInd] = clusters[realInd];
			}
		}
	}
}

__global__ void ngbrCompRO(unsigned char* img, int* clusters, int* clusterSize, int rows, int cols, float diff, int* newClust, float* clusterMeans) {

	int rowInd = blockIdx.x * blockDim.x + threadIdx.x;
	int colInd = blockIdx.y * blockDim.y + threadIdx.y;
	int realInd = rowInd * cols + colInd;



	if (rowInd < rows && colInd < cols - 1 && colInd % 2 == 1) {
		int nextInd = rowInd * cols + (colInd + 1);

		float currClustMean = clusterMeans[clusters[realInd]];

		if (abs(currClustMean - int(img[nextInd])) < diff && clusters[realInd] != clusters[nextInd]) {
			if (clusters[nextInd] = nextInd || abs(currClustMean - int(img[nextInd])) < abs(clusterMeans[clusters[nextInd]] - int(img[nextInd]))) {
				newClust[nextInd] = clusters[realInd];
			}
		}
	}
}

__global__ void ngbrCompUE(unsigned char* img, int* clusters, int* clusterSize, int rows, int cols, float diff, int* newClust, float* clusterMeans) {

	int rowInd = blockIdx.x * blockDim.x + threadIdx.x;
	int colInd = blockIdx.y * blockDim.y + threadIdx.y;
	int realInd = rowInd * cols + colInd;



	if (rowInd < rows && colInd < cols && rowInd != 0 && rowInd % 2 == 0) {
		int nextInd = (rowInd-1) * cols + (colInd);
		
		float currClustMean = clusterMeans[clusters[realInd]];
		if (abs(currClustMean - img[nextInd]) < diff && clusters[realInd] != clusters[nextInd]) {
			if (clusters[nextInd] = nextInd || abs(currClustMean - int(img[nextInd])) < abs(clusterMeans[clusters[nextInd]] - int(img[nextInd]))) {
				newClust[nextInd] = clusters[realInd];
				if (rowInd > 180 && rowInd < 200 && colInd > 180 && colInd < 200) {
					//printf("grouped");
				}
			}
			
		}
	}
}

__global__ void ngbrCompDE(unsigned char* img, int* clusters, int* clusterSize, int rows, int cols, float diff, int* newClust, float* clusterMeans) {

	int rowInd = blockIdx.x * blockDim.x + threadIdx.x;
	int colInd = blockIdx.y * blockDim.y + threadIdx.y;
	int realInd = rowInd * cols + colInd;



	if (rowInd < rows-1 && colInd < cols && rowInd % 2 == 0) {
		int nextInd = (rowInd + 1) * cols + (colInd);

		float currClustMean = clusterMeans[clusters[realInd]];
		if (abs(currClustMean - img[nextInd]) < diff && clusters[realInd] != clusters[nextInd]) {
			if (clusters[nextInd] = nextInd || abs(currClustMean - int(img[nextInd])) < abs(clusterMeans[clusters[nextInd]] - int(img[nextInd]))) {
				newClust[nextInd] = clusters[realInd];
			}
		}
	}
}

__global__ void ngbrCompUO(unsigned char* img, int* clusters, int* clusterSize, int rows, int cols, float diff, int* newClust, float* clusterMeans) {

	int rowInd = blockIdx.x * blockDim.x + threadIdx.x;
	int colInd = blockIdx.y * blockDim.y + threadIdx.y;
	int realInd = rowInd * cols + colInd;



	if (rowInd < rows && colInd < cols && rowInd != 0 && rowInd % 2 == 1) {
		int nextInd = (rowInd - 1) * cols + (colInd);

		float currClustMean = clusterMeans[clusters[realInd]];
		if (abs(currClustMean - img[nextInd]) < diff && clusters[realInd] != clusters[nextInd]) {
			if (clusters[nextInd] = nextInd || abs(currClustMean - int(img[nextInd])) < abs(clusterMeans[clusters[nextInd]] - int(img[nextInd]))) {
				newClust[nextInd] = clusters[realInd];
			}

		}
	}
}

__global__ void ngbrCompDO(unsigned char* img, int* clusters, int* clusterSize, int rows, int cols, float diff, int* newClust, float* clusterMeans) {

	int rowInd = blockIdx.x * blockDim.x + threadIdx.x;
	int colInd = blockIdx.y * blockDim.y + threadIdx.y;
	int realInd = rowInd * cols + colInd;



	if (rowInd < rows-1 && colInd < cols && rowInd % 2 == 1) {
		int nextInd = (rowInd + 1) * cols + (colInd);

		float currClustMean = clusterMeans[clusters[realInd]];
		if (abs(currClustMean - img[nextInd]) < diff && clusters[realInd] != clusters[nextInd]) {
			if (clusters[nextInd] = nextInd || abs(currClustMean - int(img[nextInd])) < abs(clusterMeans[clusters[nextInd]] - int(img[nextInd]))) {
				newClust[nextInd] = clusters[realInd];
			}
		}
	}
}

__global__ void ngbrComp(unsigned char* img, int* clusters, int* clusterSize, bool setup, int rows, int cols, float diff, int* newClust, float* clusterMeans) {

	int rowInd = blockIdx.x * blockDim.x + threadIdx.x;
	int colInd = blockIdx.y * blockDim.y + threadIdx.y;
	int realInd = rowInd * cols + colInd;

	

	if (rowInd < rows && colInd < cols) {
		float minDiff = 255.0;
		if (setup) {
			clusters[realInd] = realInd;
			clusterSize[realInd] = 1;

		}
		int nextInd;
		float currClustMean = clusterMeans[clusters[realInd]];
		/*if (rowInd > 0) {
			nextInd = (rowInd - 1) * cols + colInd;

			if (abs(currClustMean - img[nextInd]) < minDiff && clusters[realInd] != clusters[nextInd]) {
				minDiff = abs(img[realInd] - img[nextInd]);
				groupMerge = nextInd;
			}
		}
		if (rowInd < rows - 1) {
			nextInd = (rowInd + 1) * cols + colInd;

			if (abs(currClustMean - img[nextInd]) < minDiff && clusters[realInd] != clusters[nextInd]) {
				minDiff = abs(img[realInd] - img[nextInd]);
				groupMerge = nextInd;
			}
		}
		if (colInd > 0) {
			nextInd = rowInd * cols + (colInd - 1);

			if (abs(currClustMean - img[nextInd]) < minDiff && clusters[realInd] != clusters[nextInd]) {
				minDiff = abs(img[realInd] - img[nextInd]);
				groupMerge = nextInd;
			}
		}
		if (colInd < cols - 1) {
			nextInd = rowInd * cols + (colInd + 1);

			if (abs(currClustMean - img[nextInd]) < minDiff && clusters[realInd] != clusters[nextInd]) {
				minDiff = abs(img[realInd] - img[nextInd]);
				groupMerge = nextInd;
			}
		}
		
		if (abs(currClustMean - img[groupMerge]) < diff) {
			if (clusterSize[groupMerge] > clusterSize[realInd]) {
				newClust[realInd] = clusters[groupMerge];
			}else {
				newClust[groupMerge] = clusters[realInd];
			}
		}*/
		
		if (rowInd < rows - 1) {
			nextInd = (rowInd + 1) * cols + colInd;

			if (abs(currClustMean - img[nextInd]) < diff && clusters[realInd] != clusters[nextInd]) {
				newClust[nextInd] = clusters[realInd];
				minDiff = abs(currClustMean - img[nextInd]);
			}
		}
		if (colInd < cols - 1) {
			nextInd = rowInd * cols + (colInd + 1);

			if (abs(currClustMean - img[nextInd]) < diff && clusters[realInd] != clusters[nextInd]) {
				if (abs(currClustMean - img[nextInd]) < minDiff) {
					newClust[nextInd] = clusters[realInd];
				}
			}
		}
	}

}

__global__ void imOut(unsigned char* imgOut, int* clusters, float* clusterMeans, int rows, int cols) {
	int rowInd = blockIdx.x * blockDim.x + threadIdx.x;
	int colInd = blockIdx.y * blockDim.y + threadIdx.y;
	int realInd = rowInd * cols + colInd;

	if (rowInd < rows && colInd < cols) {
		imgOut[realInd] = (unsigned char)clusterMeans[clusters[realInd]];
		
	}
}

__global__ void imPProc(unsigned char* imgOut, int* clusters, float* clusterMeans, int rows, int cols, int* clusterSize, int minSize) {
	int rowInd = blockIdx.x * blockDim.x + threadIdx.x;
	int colInd = blockIdx.y * blockDim.y + threadIdx.y;
	int realInd = rowInd * cols + colInd;

	if (rowInd < rows && colInd < cols) {
		if (clusterSize[clusters[realInd]] > minSize) {
			imgOut[realInd] = (unsigned char)clusterMeans[clusters[realInd]];
		}
		else {
			imgOut[realInd] = 0;
		}
	}
}

__global__ void clusterSync(int* clusters, int rows, int cols, int* newClust, int* clusterSize, float* clustMeans) {
	int rowInd = blockIdx.x * blockDim.x + threadIdx.x;
	int colInd = blockIdx.y * blockDim.y + threadIdx.y;
	int realInd = rowInd * cols + colInd;

	if (rowInd < rows && colInd < cols) {
		clusters[realInd] = newClust[realInd];
		//clusterSize[realInd] = 0;
		//clustMeans[realInd] = 0.0;
	}
}

__global__ void clusterMeanMul(int* clusterSize, float* clusterMeans, int rows, int cols) {
	int rowInd = blockIdx.x * blockDim.x + threadIdx.x;
	int colInd = blockIdx.y * blockDim.y + threadIdx.y;
	int realInd = rowInd * cols + colInd;

	if (rowInd < rows && colInd < cols) {
		clusterMeans[realInd] = clusterMeans[realInd] * clusterSize[realInd];
	}
}

__global__ void clusterMeanSum(unsigned char* img, int* clusters, int* clusterSize, int rows, int cols, float* clusterMeans, int* newClust) {
	int rowInd = blockIdx.x * blockDim.x + threadIdx.x;
	int colInd = blockIdx.y * blockDim.y + threadIdx.y;
	int realInd = rowInd * cols + colInd;

	if (rowInd < rows && colInd < cols) {
		if (clusters[realInd] != newClust[realInd]) {
			float* address = clusterMeans + newClust[realInd];
			atomicAdd(address, float(img[realInd]));
			address = clusterMeans + clusters[realInd];
			atomicAdd(address, -1.0 * float(img[realInd]));
			int* sizeAddress = clusterSize + newClust[realInd];
			atomicAdd(sizeAddress, 1);
			sizeAddress = clusterSize + clusters[realInd];
			atomicAdd(sizeAddress, -1);
		}
		//printf("added to cluster: %d,  val: %f\n", clusters[realInd], float(img[realInd]));
		//printf("realInd: %d,  addy: %p, cluster: %d\n",  realInd, address, clusters[realInd]);
	}
}

__global__ void clusterMeanInit(int rows, int cols, float* clusterMeans, int* clusterSize, int* clusters, int* newClusters, unsigned char* img) {
	int rowInd = blockIdx.x * blockDim.x + threadIdx.x;
	int colInd = blockIdx.y * blockDim.y + threadIdx.y;
	int realInd = rowInd * cols + colInd;

	if (rowInd < rows && colInd < cols) {
		clusters[realInd] = realInd;
		//int clustid = clusters[realInd];
		newClusters[realInd] = realInd;
		clusterSize[realInd] = 0;
		int* address = clusterSize + (realInd);
		atomicAdd(address, 1);
		clusterMeans[realInd] = img[realInd];
		//printf("cluster size: %d, realInd: %d, address: %p, clusterSIze addy: %p\n", clusterSize[realInd], realInd, address, clusterSize);
		//if (rowInd > 180 && rowInd < 200 && colInd < 200 && colInd > 180) {
			//printf("img: %d", img[realInd]);
		//}
	}
}

__global__ void clusterMeanDiv(float* clusterMeans, int rows, int cols, int* clusterSize) {
	int rowInd = blockIdx.x * blockDim.x + threadIdx.x;
	int colInd = blockIdx.y * blockDim.y + threadIdx.y;
	int realInd = rowInd * cols + colInd;

	if (rowInd < rows && colInd < cols) {
		if (clusterSize[realInd] != 0) {
			clusterMeans[realInd] = clusterMeans[realInd] / clusterSize[realInd];//check if cluster size not zero
		}
		else {
			clusterMeans[realInd] = 0.0;
		}
		//printf("clusterMean: %f,  realInd: %d,  clusterSize: %d\n", clusterMeans[realInd], realInd, clusterSize[realInd]);
		//if (clusterSize[realInd] > 10) {
			//printf("largeCluster\n");
		//}
	}
}

__global__ void pullMerge(unsigned char* img, int* clusters, int rows, int cols, float diff, int* newClust, float* clusterMeans, bool offset) {
	int rowInd = blockIdx.x * blockDim.x + threadIdx.x;
	int colInd = blockIdx.y * blockDim.y + threadIdx.y;
	int realInd = rowInd * cols + colInd;

	if (rowInd < rows && colInd < cols) {
		int bestClust = -1;
		float bestDiff = 255;
		if (offset && realInd %2 == 1) {
			if (rowInd != 0) {
				int nextInd = (rowInd - 1) * cols + colInd;
				if (abs(clusterMeans[clusters[nextInd]] - img[realInd]) < bestDiff) {
					bestDiff = abs(clusterMeans[clusters[nextInd]] - img[realInd]);
					bestClust = clusters[nextInd];
				}
			}
			if (rowInd < rows - 1) {
				int nextInd = (rowInd + 1) * cols + colInd;
				if (abs(clusterMeans[clusters[nextInd]] - img[realInd]) < bestDiff) {
					bestDiff = abs(clusterMeans[clusters[nextInd]] - img[realInd]);
					bestClust = clusters[nextInd];
				}
			}
			if (colInd != 0) {
				int nextInd = rowInd * cols + cols - 1;
				if (abs(clusterMeans[clusters[nextInd]] - img[realInd]) < bestDiff) {
					bestDiff = abs(clusterMeans[clusters[nextInd]] - img[realInd]);
					bestClust = clusters[nextInd];
				}
			}
			if (colInd < cols - 1) {
				int nextInd = rowInd * cols + cols + 1;
				if (abs(clusterMeans[clusters[nextInd]] - img[realInd]) < bestDiff) {
					bestDiff = abs(clusterMeans[clusters[nextInd]] - img[realInd]);
					bestClust = clusters[nextInd];
				}
			}
		}
		else if (!offset && realInd % 2 == 0) {
			if (rowInd != 0) {
				int nextInd = (rowInd - 1) * cols + colInd;
				if (abs(clusterMeans[clusters[nextInd]] - img[realInd]) < bestDiff) {
					bestDiff = abs(clusterMeans[clusters[nextInd]] - img[realInd]);
					bestClust = clusters[nextInd];
				}
			}
			if (rowInd < rows - 1) {
				int nextInd = (rowInd + 1) * cols + colInd;
				if (abs(clusterMeans[clusters[nextInd]] - img[realInd]) < bestDiff) {
					bestDiff = abs(clusterMeans[clusters[nextInd]] - img[realInd]);
					bestClust = clusters[nextInd];
				}
			}
			if (colInd != 0) {
				int nextInd = rowInd * cols + cols - 1;
				if (abs(clusterMeans[clusters[nextInd]] - img[realInd]) < bestDiff) {
					bestDiff = abs(clusterMeans[clusters[nextInd]] - img[realInd]);
					bestClust = clusters[nextInd];
				}
			}
			if (colInd < cols - 1) {
				int nextInd = rowInd * cols + cols + 1;
				if (abs(clusterMeans[clusters[nextInd]] - img[realInd]) < bestDiff) {
					bestDiff = abs(clusterMeans[clusters[nextInd]] - img[realInd]);
					bestClust = clusters[nextInd];
				}
			}
		}
		newClust[realInd] = bestClust;
	}
}

__global__ void clusterIsolateKernel(unsigned char* Rchan, unsigned char* Gchan, unsigned char* Bchan, int* clusterIDS, int rows, int cols, char R, char G, char B, int clusterNum) {
	int rowInd = blockIdx.x * blockDim.x + threadIdx.x;
	int colInd = blockIdx.y * blockDim.y + threadIdx.y;
	int realInd = rowInd * cols + colInd;

	if (rowInd < rows && colInd < cols) {
		if (clusterIDS[realInd] == clusterNum) {
			Rchan[realInd] = R;
			Gchan[realInd] = G;
			Bchan[realInd] = B;
		}
	}
}


__global__ void clusterBoundaryDraw(unsigned char* d_img, int* clusters, int rows, int cols, float* clustMeans, float diff, bool allSeg) {
	int rowInd = blockIdx.x * blockDim.x + threadIdx.x;
	int colInd = blockIdx.y * blockDim.y + threadIdx.y;
	int realInd = rowInd * cols + colInd;

	if (rowInd < rows && colInd < cols) {
		bool border = false;
		if (allSeg) {
			if (rowInd < rows - 1) {
				int nextInd = (rowInd + 1) * cols + (colInd);
				if (clusters[nextInd] != clusters[realInd]) {
					border = true;
				}
			}
			if (rowInd != 0) {
				int nextInd = (rowInd - 1) * cols + (colInd);
				if (clusters[nextInd] != clusters[realInd]) {
					border = true;
				}
			}
			if (colInd < cols - 1) {
				int nextInd = (rowInd) * cols + (colInd+1);
				if (clusters[nextInd] != clusters[realInd]) {
					border = true;
				}
			}
			if (colInd != 0) {
				int nextInd = (rowInd) * cols + (colInd-1);
				if (clusters[nextInd] != clusters[realInd]) {
					border = true;
				}
			}
		}
		else {
			if (rowInd < rows - 1) {
				int nextInd = (rowInd + 1) * cols + (colInd);
				if (clusters[nextInd] != clusters[realInd] && abs(clustMeans[clusters[nextInd]] - clustMeans[clusters[realInd]]) > diff) {
					border = true;
				}
			}
			if (rowInd != 0) {
				int nextInd = (rowInd - 1) * cols + (colInd);
				if (clusters[nextInd] != clusters[realInd] && abs(clustMeans[clusters[nextInd]] - clustMeans[clusters[realInd]]) > diff) {
					border = true;
				}
			}
			if (colInd < cols - 1) {
				int nextInd = (rowInd)*cols + (colInd + 1);
				if (clusters[nextInd] != clusters[realInd] && abs(clustMeans[clusters[nextInd]] - clustMeans[clusters[realInd]]) > diff) {
					border = true;
				}
			}
			if (colInd != 0) {
				int nextInd = (rowInd)*cols + (colInd - 1);
				if (clusters[nextInd] != clusters[realInd] && abs(clustMeans[clusters[nextInd]] - clustMeans[clusters[realInd]]) > diff) {
					border = true;
				}
			}
		}
		if (border) {
			d_img[realInd] = 0;
		}
		else {
			d_img[realInd] = 255;
		}
	}
}
using namespace cv;

namespace seg {

	Mat clusterIsolate(Mat input, int* clusterIDs, char R, char G, char B, int ClusterNum) {
		std::vector<uchar> RmVec((input.rows * input.cols), 0);
		std::vector<uchar> GmVec((input.rows * input.cols), 0);
		std::vector<uchar> BmVec((input.rows * input.cols), 0);
		//if (inputBW.isContinuous()) {
			//mVec.assign(inputBW.data, inputBW.data + inputBW.total());
		//}
		//else {
		for (int r = 0; r < input.rows; r++) {
			for (int c = 0; c < input.cols; c++) {
				int index = input.cols * r + c;
				RmVec[index] = input.at<uchar>(r, c);
				GmVec[index] = input.at<uchar>(r, c);
				BmVec[index] = input.at<uchar>(r, c);
			}
		}
		unsigned char* d_imgR, * d_imgG,* d_imgB;
		
		int * d_clusterIDs;
		int size = sizeof(unsigned char) * input.rows * input.cols;
		hipMalloc((void**)&d_imgR, size); hipMalloc((void**)&d_imgG, size); hipMalloc((void**)&d_imgB, size);
		hipMemcpy(d_imgR, (unsigned char*)&RmVec[0], size, hipMemcpyHostToDevice); hipMemcpy(d_imgG, (unsigned char*)&GmVec[0], size, hipMemcpyHostToDevice); hipMemcpy(d_imgB, (unsigned char*)&BmVec[0], size, hipMemcpyHostToDevice);
		size = sizeof(int) * input.rows * input.cols;
		hipMalloc((void**)&d_clusterIDs, size); 
		hipMemcpy(d_clusterIDs, clusterIDs, size, hipMemcpyHostToDevice);
		
		
		//printf("variance: %f,  diff: %f\n", imStat.first, diff);
		int numBlocksH = input.cols / BLOCK_SIZE;
		if (input.cols % BLOCK_SIZE > 0) {
			numBlocksH++;
		}
		int numBlocksV = input.rows / BLOCK_SIZE;
		if (input.rows % BLOCK_SIZE > 0) {
			numBlocksV++;
		}
		dim3 gGrid(numBlocksV, numBlocksH);
		dim3 bGrid(BLOCK_SIZE, BLOCK_SIZE);
		printf("Kernel Call \n");
		clusterIsolateKernel << <gGrid, bGrid >> > (d_imgR, d_imgG, d_imgB, d_clusterIDs, input.rows, input.cols, R, G, B, ClusterNum);
		printf("Isolated \n");
		hipFree(d_clusterIDs);
		size = sizeof(char) * input.rows * input.cols;
		hipMemcpy((unsigned char*)&RmVec[0], d_imgR,  size, hipMemcpyDeviceToHost); hipMemcpy((unsigned char*)&GmVec[0], d_imgG,  size, hipMemcpyDeviceToHost); hipMemcpy((unsigned char*)&BmVec[0], d_imgB,  size, hipMemcpyDeviceToHost);
		hipFree(d_imgR); hipFree(d_imgG); hipFree(d_imgB);
		Mat retImg = Mat(input.rows, input.cols, CV_8UC3);
		for (int r = 0; r < input.rows; r++) {
			for (int c = 0; c < input.cols; c++) {
				int index = input.cols * r + c;
				retImg.at<Vec3b>(r, c).val[0] = RmVec[index];
				retImg.at<Vec3b>(r, c).val[1] = GmVec[index];
				retImg.at<Vec3b>(r, c).val[2] = BmVec[index];
			}
		}
		printf("Returning \n");
		return retImg;
	}

	Mat laplaceEdge(cv::Mat inputBW) {

		std::vector<uchar> mVec((inputBW.rows * inputBW.cols), 0);
		//if (inputBW.isContinuous()) {
			//mVec.assign(inputBW.data, inputBW.data + inputBW.total());
		//}
		//else {
		for (int r = 0; r < inputBW.rows; r++) {
			for (int c = 0; c < inputBW.cols; c++) {
				int index = inputBW.cols * r + c;
				mVec[index] = inputBW.at<uchar>(r, c);
			}
		}
		//}

		unsigned char* d_img, * d_imgN;
		int size = sizeof(unsigned char) * inputBW.rows * inputBW.cols;
		hipMalloc((void**)&d_img, size); hipMalloc((void**)&d_imgN, size);
		hipMemcpy(d_img, (unsigned char*)&mVec[0], size, hipMemcpyHostToDevice);

		int numBlocksH = inputBW.cols / BLOCK_SIZE;
		if (inputBW.cols % BLOCK_SIZE > 0) {
			numBlocksH++;
		}
		int numBlocksV = inputBW.rows / BLOCK_SIZE;
		if (inputBW.rows % BLOCK_SIZE > 0) {
			numBlocksV++;
		}
		dim3 gGrid(numBlocksV, numBlocksH);
		dim3 bGrid(BLOCK_SIZE, BLOCK_SIZE);

		//printf("row blocks: %d,  col blocks: %d\n", numBlocksV, numBlocksH);
		laplace << <gGrid, bGrid >> > (d_img, d_imgN, inputBW.rows, inputBW.cols);

		hipMemcpy((unsigned char*)&mVec[0], d_imgN, size, hipMemcpyDeviceToHost);
		hipFree(d_img); hipFree(d_imgN);
		for (int r = 0; r < inputBW.rows; r++) {
			for (int c = 0; c < inputBW.cols; c++) {
				int index = (inputBW.cols * r) + c;

				inputBW.at<uchar>(r, c) = mVec[index];
			}
		}
		return inputBW;
	}


	float mean(cv::Mat input) {

		uint8_t* imgptr;
		float* rowMeans;
		//printf("gpumat start\n");
		//cuda::GpuMat gpuInput(sizeof(unsigned char)*input.rows*input.cols, CV_8UC1);
		cuda::GpuMat gpuInput(input.rows, input.cols, CV_8UC1);
		//printf("gpumat created\n");

		gpuInput.upload(input);
		hipMalloc((void**)&imgptr, sizeof(unsigned char) * input.rows * input.cols);
		hipMalloc((void**)&rowMeans, sizeof(float) * input.rows);

		hipMemcpy(imgptr, gpuInput.ptr<uint8_t>(), sizeof(unsigned char) * input.rows * input.cols, hipMemcpyDeviceToDevice);
		//printf("gpumat sent to cudamem\n");
		int numBlocks = input.rows / BLOCK_SIZE;
		if (input.rows % BLOCK_SIZE > 0) {
			numBlocks++;
		}
		meanIMG << <numBlocks, BLOCK_SIZE >> > (imgptr, input.rows, input.cols, rowMeans);
		std::vector<float> rowMeansC(input.rows, 0.0);
		hipMemcpy((float*)&rowMeansC[0], rowMeans, sizeof(float) * input.rows, hipMemcpyDeviceToHost);
		printf("successful vector copy\n");
		float totMean = 0.0;
		for (int i = 0; i < rowMeansC.size(); i++) {
			totMean += rowMeansC[i];
		}
		totMean /= rowMeansC.size();

		hipFree(imgptr); hipFree(rowMeans);
		
		return totMean;
	}
	
	std::pair<float,float> variance(Mat input) {
		uint8_t* imgptr;
		float* rowMeans;
		
		cuda::GpuMat gpuInput(input.rows, input.cols, CV_8UC1);
		

		gpuInput.upload(input);
		hipMalloc((void**)&imgptr, sizeof(unsigned char) * input.rows * input.cols);
		hipMalloc((void**)&rowMeans, sizeof(float) * input.rows);

		hipMemcpy(imgptr, gpuInput.ptr<uint8_t>(), sizeof(unsigned char) * input.rows * input.cols, hipMemcpyDeviceToDevice);
		//printf("gpumat sent to cudamem\n");
		int numBlocks = input.rows / BLOCK_SIZE;
		if (input.rows % BLOCK_SIZE > 0) {
			numBlocks++;
		}
		meanIMG << <numBlocks, BLOCK_SIZE >> > (imgptr, input.rows, input.cols, rowMeans);
		std::vector<float> rowMeansC(input.rows, 0.0);
		hipMemcpy((float*)&rowMeansC[0], rowMeans, sizeof(float) * input.rows, hipMemcpyDeviceToHost);
		printf("successful vector copy\n");
		float totMean = 0.0;
		for (int i = 0; i < rowMeansC.size(); i++) {
			totMean += rowMeansC[i];
		}
		totMean /= rowMeansC.size();

		hipFree(rowMeans);

		float* varSums; std::vector<float> varSumsC(input.rows, 0.0);
		hipMalloc((void**)&varSums, sizeof(float) * input.rows);
		varIMG << <numBlocks, BLOCK_SIZE >> > (imgptr, input.rows, input.cols, totMean, varSums);
		hipMemcpy((float*)&varSumsC[0], varSums, sizeof(float) * input.rows, hipMemcpyDeviceToHost);

		float varSum = 0.0;
		for (auto val : varSumsC) {
			varSum += val;
		}
		varSum /= varSumsC.size();
		hipFree(imgptr); hipFree(varSums);
		return std::make_pair(varSum, totMean);
	}

	std::pair<float, float> varNOMEM(uint8_t* imgptr, int rows, int cols) {
		float* rowMeans;
		int numBlocks =rows / BLOCK_SIZE;
		if (rows % BLOCK_SIZE > 0) {
			numBlocks++;
		}
		hipMalloc((void**)&rowMeans, sizeof(float) * rows);
		meanIMG << <numBlocks, BLOCK_SIZE >> > (imgptr, rows, cols, rowMeans);
		std::vector<float> rowMeansC(rows, 0.0);
		hipMemcpy((float*)&rowMeansC[0], rowMeans, sizeof(float) * rows, hipMemcpyDeviceToHost);
		//printf("successful vector copy\n");
		float totMean = 0.0;
		for (int i = 0; i < rowMeansC.size(); i++) {
			totMean += rowMeansC[i];
		}
		totMean /= rowMeansC.size();

		hipFree(rowMeans);

		float* varSums; std::vector<float> varSumsC(rows, 0.0);
		hipMalloc((void**)&varSums, sizeof(float) * rows);
		varIMG << <numBlocks, BLOCK_SIZE >> > (imgptr, rows, cols, totMean, varSums);
		hipMemcpy((float*)&varSumsC[0], varSums, sizeof(float) * rows, hipMemcpyDeviceToHost);
		hipFree(varSums);
		float varSum = 0.0;
		for (auto val : varSumsC) {
			varSum += val;
		}
		varSum /= varSumsC.size();
		return std::make_pair(varSum, totMean);
	}

	std::pair<Mat, int*> segment(Mat input, float diffParam, int cycles) {
		std::vector<uchar> mVec((input.rows * input.cols), 0);
		//if (inputBW.isContinuous()) {
			//mVec.assign(inputBW.data, inputBW.data + inputBW.total());
		//}
		//else {
		for (int r = 0; r < input.rows; r++) {
			for (int c = 0; c < input.cols; c++) {
				int index = input.cols * r + c;
				mVec[index] = input.at<uchar>(r, c);
			}
		}
		//}

		unsigned char* d_img;
		float* clustMean;
		int* clusterSize, * clusterIDs, * newIDs;
		int size = sizeof(unsigned char) * input.rows * input.cols;
		hipMalloc((void**)&d_img, size);
		hipMemcpy(d_img, (unsigned char*)&mVec[0], size, hipMemcpyHostToDevice);
		size = sizeof(int) * input.rows * input.cols;
		hipMalloc((void**)&clusterSize, size); hipMalloc((void**)&clusterIDs, size); hipMalloc((void**)&newIDs, size);
		size = sizeof(float) * input.rows * input.cols;
		hipMalloc((void**)&clustMean, size);
		std::pair<float, float> imStat = varNOMEM(d_img, input.rows, input.cols);
		float diff = imStat.first * diffParam;
		//printf("variance: %f,  diff: %f\n", imStat.first, diff);
		int numBlocksH = input.cols / BLOCK_SIZE;
		if (input.cols % BLOCK_SIZE > 0) {
			numBlocksH++;
		}
		int numBlocksV = input.rows / BLOCK_SIZE;
		if (input.rows % BLOCK_SIZE > 0) {
			numBlocksV++;
		}
		dim3 gGrid(numBlocksV, numBlocksH);
		dim3 bGrid(BLOCK_SIZE, BLOCK_SIZE);

		clusterMeanInit << <gGrid, bGrid >> > (input.rows, input.cols, clustMean, clusterSize, clusterIDs, newIDs, d_img);
		/*
		bool vec:
		0: LE, 1: RE, 2: LO, 3: RO, 4: UE, 5: DE, 6: UO, 7: DO
		*/
		bool altRandDet = true;
		int altRandDet_detcount = 10;
		int alrRandDet_randcount = 20;
		bool random = false;
		bool postProc = false;
		int minSize = 10;
		bool altMerge = false;
		bool modOrder = false;
		float diffMod = 0.97;
		float oldDiff = diffParam;
		bool pullMergeKern = true;
		std::vector<bool> segVec(12, false);
		if (!pullMergeKern) {
			for (int i = 0; i < cycles; i++) {
				if (altRandDet) {
					if (i % (altRandDet_detcount + alrRandDet_randcount) < alrRandDet_randcount) {
						random = true;
					}
					else {
						random = false;
					}
				}
				//diffParam *= diffMod;
				if (diffParam < oldDiff / 5.0) {
					//diffParam = oldDiff;
				}
				if (random) {
					for (int j = 0; j < 8; j++) {
						segVec[j] = rand() % 3 == 1;
					}
					if (segVec[0]) {
						ngbrCompLE << < gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, diff, newIDs, clustMean);


						clusterMeanMul << <gGrid, bGrid >> > (clusterSize, clustMean, input.rows, input.cols);
						clusterMeanSum << <gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, clustMean, newIDs);
						clusterMeanDiv << <gGrid, bGrid >> > (clustMean, input.rows, input.cols, clusterSize);
						clusterSync << <gGrid, bGrid >> > (clusterIDs, input.rows, input.cols, newIDs, clusterSize, clustMean);
					}
					if (segVec[1]) {
						ngbrCompRE << < gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, diff, newIDs, clustMean);


						clusterMeanMul << <gGrid, bGrid >> > (clusterSize, clustMean, input.rows, input.cols);
						clusterMeanSum << <gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, clustMean, newIDs);
						clusterMeanDiv << <gGrid, bGrid >> > (clustMean, input.rows, input.cols, clusterSize);
						clusterSync << <gGrid, bGrid >> > (clusterIDs, input.rows, input.cols, newIDs, clusterSize, clustMean);
					}
					if (segVec[4]) {
						ngbrCompUE << < gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, diff, newIDs, clustMean);


						clusterMeanMul << <gGrid, bGrid >> > (clusterSize, clustMean, input.rows, input.cols);
						clusterMeanSum << <gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, clustMean, newIDs);
						clusterMeanDiv << <gGrid, bGrid >> > (clustMean, input.rows, input.cols, clusterSize);
						clusterSync << <gGrid, bGrid >> > (clusterIDs, input.rows, input.cols, newIDs, clusterSize, clustMean);
					}
					if (segVec[5]) {
						ngbrCompDE << < gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, diff, newIDs, clustMean);


						clusterMeanMul << <gGrid, bGrid >> > (clusterSize, clustMean, input.rows, input.cols);
						clusterMeanSum << <gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, clustMean, newIDs);
						clusterMeanDiv << <gGrid, bGrid >> > (clustMean, input.rows, input.cols, clusterSize);
						clusterSync << <gGrid, bGrid >> > (clusterIDs, input.rows, input.cols, newIDs, clusterSize, clustMean);
					}
					if (segVec[2]) {
						ngbrCompLO << < gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, diff, newIDs, clustMean);


						clusterMeanMul << <gGrid, bGrid >> > (clusterSize, clustMean, input.rows, input.cols);
						clusterMeanSum << <gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, clustMean, newIDs);
						clusterMeanDiv << <gGrid, bGrid >> > (clustMean, input.rows, input.cols, clusterSize);
						clusterSync << <gGrid, bGrid >> > (clusterIDs, input.rows, input.cols, newIDs, clusterSize, clustMean);
					}
					if (segVec[3]) {
						ngbrCompRO << < gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, diff, newIDs, clustMean);


						clusterMeanMul << <gGrid, bGrid >> > (clusterSize, clustMean, input.rows, input.cols);
						clusterMeanSum << <gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, clustMean, newIDs);
						clusterMeanDiv << <gGrid, bGrid >> > (clustMean, input.rows, input.cols, clusterSize);
						clusterSync << <gGrid, bGrid >> > (clusterIDs, input.rows, input.cols, newIDs, clusterSize, clustMean);
					}
					if (segVec[6]) {
						ngbrCompUO << < gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, diff, newIDs, clustMean);


						clusterMeanMul << <gGrid, bGrid >> > (clusterSize, clustMean, input.rows, input.cols);
						clusterMeanSum << <gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, clustMean, newIDs);
						clusterMeanDiv << <gGrid, bGrid >> > (clustMean, input.rows, input.cols, clusterSize);
						clusterSync << <gGrid, bGrid >> > (clusterIDs, input.rows, input.cols, newIDs, clusterSize, clustMean);
					}
					if (segVec[7]) {
						ngbrCompDO << < gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, diff, newIDs, clustMean);

						clusterMeanMul << <gGrid, bGrid >> > (clusterSize, clustMean, input.rows, input.cols);
						clusterMeanSum << <gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, clustMean, newIDs);
						clusterMeanDiv << <gGrid, bGrid >> > (clustMean, input.rows, input.cols, clusterSize);
						clusterSync << <gGrid, bGrid >> > (clusterIDs, input.rows, input.cols, newIDs, clusterSize, clustMean);

					}
					if (segVec[8]) {
						ngbrCompLUE << < gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, diff, newIDs, clustMean);


						clusterMeanMul << <gGrid, bGrid >> > (clusterSize, clustMean, input.rows, input.cols);
						clusterMeanSum << <gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, clustMean, newIDs);
						clusterMeanDiv << <gGrid, bGrid >> > (clustMean, input.rows, input.cols, clusterSize);
						clusterSync << <gGrid, bGrid >> > (clusterIDs, input.rows, input.cols, newIDs, clusterSize, clustMean);
					}
					if (segVec[9]) {
						ngbrCompLDE << < gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, diff, newIDs, clustMean);


						clusterMeanMul << <gGrid, bGrid >> > (clusterSize, clustMean, input.rows, input.cols);
						clusterMeanSum << <gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, clustMean, newIDs);
						clusterMeanDiv << <gGrid, bGrid >> > (clustMean, input.rows, input.cols, clusterSize);
						clusterSync << <gGrid, bGrid >> > (clusterIDs, input.rows, input.cols, newIDs, clusterSize, clustMean);
					}
					if (segVec[10]) {
						ngbrCompRUE << < gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, diff, newIDs, clustMean);


						clusterMeanMul << <gGrid, bGrid >> > (clusterSize, clustMean, input.rows, input.cols);
						clusterMeanSum << <gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, clustMean, newIDs);
						clusterMeanDiv << <gGrid, bGrid >> > (clustMean, input.rows, input.cols, clusterSize);
						clusterSync << <gGrid, bGrid >> > (clusterIDs, input.rows, input.cols, newIDs, clusterSize, clustMean);
					}
					if (segVec[11]) {
						ngbrCompRDE << < gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, diff, newIDs, clustMean);


						clusterMeanMul << <gGrid, bGrid >> > (clusterSize, clustMean, input.rows, input.cols);
						clusterMeanSum << <gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, clustMean, newIDs);
						clusterMeanDiv << <gGrid, bGrid >> > (clustMean, input.rows, input.cols, clusterSize);
						clusterSync << <gGrid, bGrid >> > (clusterIDs, input.rows, input.cols, newIDs, clusterSize, clustMean);
					}
				}
				/*else if (altMerge) {
					ngbrCompLEA << < gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, diff, newIDs, clustMean);

					clusterSync << <gGrid, bGrid >> > (clusterIDs, input.rows, input.cols, newIDs, clusterSize, clustMean);
					clusterMeanSum << <gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, clustMean);
					hipDeviceSynchronize();
					clusterMeanDiv << <gGrid, bGrid >> > (clustMean, input.rows, input.cols, clusterSize);


					ngbrCompREA << < gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, diff, newIDs, clustMean);

					clusterSync << <gGrid, bGrid >> > (clusterIDs, input.rows, input.cols, newIDs, clusterSize, clustMean);

					clusterMeanSum << <gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, clustMean);
					hipDeviceSynchronize();
					clusterMeanDiv << <gGrid, bGrid >> > (clustMean, input.rows, input.cols, clusterSize);

					ngbrCompUEA << < gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, diff, newIDs, clustMean);

					clusterSync << <gGrid, bGrid >> > (clusterIDs, input.rows, input.cols, newIDs, clusterSize, clustMean);

					clusterMeanSum << <gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, clustMean);
					hipDeviceSynchronize();
					clusterMeanDiv << <gGrid, bGrid >> > (clustMean, input.rows, input.cols, clusterSize);

					ngbrCompDEA << < gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, diff, newIDs, clustMean);

					clusterSync << <gGrid, bGrid >> > (clusterIDs, input.rows, input.cols, newIDs, clusterSize, clustMean);

					clusterMeanSum << <gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, clustMean);
					hipDeviceSynchronize();
					clusterMeanDiv << <gGrid, bGrid >> > (clustMean, input.rows, input.cols, clusterSize);

					ngbrCompLOA << < gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, diff, newIDs, clustMean);

					clusterSync << <gGrid, bGrid >> > (clusterIDs, input.rows, input.cols, newIDs, clusterSize, clustMean);

					clusterMeanSum << <gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, clustMean);
					hipDeviceSynchronize();
					clusterMeanDiv << <gGrid, bGrid >> > (clustMean, input.rows, input.cols, clusterSize);

					ngbrCompROA << < gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, diff, newIDs, clustMean);

					clusterSync << <gGrid, bGrid >> > (clusterIDs, input.rows, input.cols, newIDs, clusterSize, clustMean);

					clusterMeanSum << <gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, clustMean);
					hipDeviceSynchronize();
					clusterMeanDiv << <gGrid, bGrid >> > (clustMean, input.rows, input.cols, clusterSize);

					ngbrCompUOA << < gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, diff, newIDs, clustMean);

					clusterSync << <gGrid, bGrid >> > (clusterIDs, input.rows, input.cols, newIDs, clusterSize, clustMean);

					clusterMeanSum << <gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, clustMean);
					hipDeviceSynchronize();
					clusterMeanDiv << <gGrid, bGrid >> > (clustMean, input.rows, input.cols, clusterSize);

					ngbrCompDOA << < gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, diff, newIDs, clustMean);

					clusterSync << <gGrid, bGrid >> > (clusterIDs, input.rows, input.cols, newIDs, clusterSize, clustMean);

					clusterMeanSum << <gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, clustMean);
					hipDeviceSynchronize();
					clusterMeanDiv << <gGrid, bGrid >> > (clustMean, input.rows, input.cols, clusterSize);
				}//
				/*else if (!modOrder) {
						ngbrCompLE << < gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, diff, newIDs, clustMean);

						clusterSync << <gGrid, bGrid >> > (clusterIDs, input.rows, input.cols, newIDs, clusterSize, clustMean);
						clusterMeanSum << <gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, clustMean);
						hipDeviceSynchronize();
						clusterMeanDiv << <gGrid, bGrid >> > (clustMean, input.rows, input.cols, clusterSize);


						ngbrCompRE << < gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, diff, newIDs, clustMean);

						clusterSync << <gGrid, bGrid >> > (clusterIDs, input.rows, input.cols, newIDs, clusterSize, clustMean);

						clusterMeanSum << <gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, clustMean);
						hipDeviceSynchronize();
						clusterMeanDiv << <gGrid, bGrid >> > (clustMean, input.rows, input.cols, clusterSize);

						ngbrCompUE << < gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, diff, newIDs, clustMean);

						clusterSync << <gGrid, bGrid >> > (clusterIDs, input.rows, input.cols, newIDs, clusterSize, clustMean);

						clusterMeanSum << <gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, clustMean);
						hipDeviceSynchronize();
						clusterMeanDiv << <gGrid, bGrid >> > (clustMean, input.rows, input.cols, clusterSize);

						ngbrCompDE << < gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, diff, newIDs, clustMean);

						clusterSync << <gGrid, bGrid >> > (clusterIDs, input.rows, input.cols, newIDs, clusterSize, clustMean);

						clusterMeanSum << <gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, clustMean);
						hipDeviceSynchronize();
						clusterMeanDiv << <gGrid, bGrid >> > (clustMean, input.rows, input.cols, clusterSize);

						ngbrCompLO << < gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, diff, newIDs, clustMean);

						clusterSync << <gGrid, bGrid >> > (clusterIDs, input.rows, input.cols, newIDs, clusterSize, clustMean);

						clusterMeanSum << <gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, clustMean);
						hipDeviceSynchronize();
						clusterMeanDiv << <gGrid, bGrid >> > (clustMean, input.rows, input.cols, clusterSize);

						ngbrCompRO << < gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, diff, newIDs, clustMean);

						clusterSync << <gGrid, bGrid >> > (clusterIDs, input.rows, input.cols, newIDs, clusterSize, clustMean);

						clusterMeanSum << <gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, clustMean);
						hipDeviceSynchronize();
						clusterMeanDiv << <gGrid, bGrid >> > (clustMean, input.rows, input.cols, clusterSize);

						ngbrCompUO << < gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, diff, newIDs, clustMean);

						clusterSync << <gGrid, bGrid >> > (clusterIDs, input.rows, input.cols, newIDs, clusterSize, clustMean);

						clusterMeanSum << <gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, clustMean);
						hipDeviceSynchronize();
						clusterMeanDiv << <gGrid, bGrid >> > (clustMean, input.rows, input.cols, clusterSize);

						ngbrCompDO << < gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, diff, newIDs, clustMean);

						clusterSync << <gGrid, bGrid >> > (clusterIDs, input.rows, input.cols, newIDs, clusterSize, clustMean);

						clusterMeanSum << <gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, clustMean);
						hipDeviceSynchronize();
						clusterMeanDiv << <gGrid, bGrid >> > (clustMean, input.rows, input.cols, clusterSize);

				}*/
				else {

					ngbrCompRE << < gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, diff, newIDs, clustMean);


					clusterMeanMul << <gGrid, bGrid >> > (clusterSize, clustMean, input.rows, input.cols);
					clusterMeanSum << <gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, clustMean, newIDs);
					clusterMeanDiv << <gGrid, bGrid >> > (clustMean, input.rows, input.cols, clusterSize);
					clusterSync << <gGrid, bGrid >> > (clusterIDs, input.rows, input.cols, newIDs, clusterSize, clustMean);
					ngbrCompLE << < gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, diff, newIDs, clustMean);


					clusterMeanMul << <gGrid, bGrid >> > (clusterSize, clustMean, input.rows, input.cols);
					clusterMeanSum << <gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, clustMean, newIDs);
					clusterMeanDiv << <gGrid, bGrid >> > (clustMean, input.rows, input.cols, clusterSize);
					clusterSync << <gGrid, bGrid >> > (clusterIDs, input.rows, input.cols, newIDs, clusterSize, clustMean);
					ngbrCompDE << < gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, diff, newIDs, clustMean);


					clusterMeanMul << <gGrid, bGrid >> > (clusterSize, clustMean, input.rows, input.cols);
					clusterMeanSum << <gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, clustMean, newIDs);
					clusterMeanDiv << <gGrid, bGrid >> > (clustMean, input.rows, input.cols, clusterSize);
					clusterSync << <gGrid, bGrid >> > (clusterIDs, input.rows, input.cols, newIDs, clusterSize, clustMean);
					ngbrCompUE << < gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, diff, newIDs, clustMean);


					clusterMeanMul << <gGrid, bGrid >> > (clusterSize, clustMean, input.rows, input.cols);
					clusterMeanSum << <gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, clustMean, newIDs);
					clusterMeanDiv << <gGrid, bGrid >> > (clustMean, input.rows, input.cols, clusterSize);
					clusterSync << <gGrid, bGrid >> > (clusterIDs, input.rows, input.cols, newIDs, clusterSize, clustMean);
					ngbrCompRO << < gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, diff, newIDs, clustMean);


					clusterMeanMul << <gGrid, bGrid >> > (clusterSize, clustMean, input.rows, input.cols);
					clusterMeanSum << <gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, clustMean, newIDs);
					clusterMeanDiv << <gGrid, bGrid >> > (clustMean, input.rows, input.cols, clusterSize);
					clusterSync << <gGrid, bGrid >> > (clusterIDs, input.rows, input.cols, newIDs, clusterSize, clustMean);
					ngbrCompLO << < gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, diff, newIDs, clustMean);


					clusterMeanMul << <gGrid, bGrid >> > (clusterSize, clustMean, input.rows, input.cols);
					clusterMeanSum << <gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, clustMean, newIDs);
					clusterMeanDiv << <gGrid, bGrid >> > (clustMean, input.rows, input.cols, clusterSize);
					clusterSync << <gGrid, bGrid >> > (clusterIDs, input.rows, input.cols, newIDs, clusterSize, clustMean);
					ngbrCompDO << < gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, diff, newIDs, clustMean);


					clusterMeanMul << <gGrid, bGrid >> > (clusterSize, clustMean, input.rows, input.cols);
					clusterMeanSum << <gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, clustMean, newIDs);
					clusterMeanDiv << <gGrid, bGrid >> > (clustMean, input.rows, input.cols, clusterSize);
					clusterSync << <gGrid, bGrid >> > (clusterIDs, input.rows, input.cols, newIDs, clusterSize, clustMean);
					ngbrCompUO << < gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, diff, newIDs, clustMean);


					clusterMeanMul << <gGrid, bGrid >> > (clusterSize, clustMean, input.rows, input.cols);
					clusterMeanSum << <gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, clustMean, newIDs);
					clusterMeanDiv << <gGrid, bGrid >> > (clustMean, input.rows, input.cols, clusterSize);
					clusterSync << <gGrid, bGrid >> > (clusterIDs, input.rows, input.cols, newIDs, clusterSize, clustMean);

				}
			}
		}else {
			for (int i = 0; i < cycles; i++) {
				bool offset = i % 2 == 0;

				pullMerge << <gGrid, bGrid >> > (d_img, clusterIDs input.rows, input.cols, diff, newIDs, clustMean, offset);
				clusterMeanMul << <gGrid, bGrid >> > (clusterSize, clustMean, input.rows, input.cols);
				clusterMeanSum << <gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, clustMean, newIDs);
				clusterMeanDiv << <gGrid, bGrid >> > (clustMean, input.rows, input.cols, clusterSize);
				clusterSync << <gGrid, bGrid >> > (clusterIDs, input.rows, input.cols, newIDs, clusterSize, clustMean);
			}
		}
		//printf("segment iter:%d \n", i);
		//clusterSync << <gGrid, bGrid >> > (clusterIDs, input.rows, input.cols, newIDs, clusterSize, clustMean);
		//clusterMeanSum << <gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, clustMean);
		//clusterMeanDiv << <gGrid, bGrid >> > (clustMean, input.rows, input.cols, clusterSize);
		if (postProc) {
			imPProc << <gGrid, bGrid >> > (d_img, clusterIDs, clustMean, input.rows, input.cols, clusterSize, minSize);
		}
		else {
			imOut << <gGrid, bGrid >> > (d_img, clusterIDs, clustMean, input.rows, input.cols);
		}
		bool boundaryDraw = false;
		if (boundaryDraw) {
			clusterBoundaryDraw << <gGrid, bGrid >> > (d_img, clusterIDs, input.rows, input.cols, clustMean, diff, true);
		}
		int* outClusters = new int[input.rows * input.cols];
		size = sizeof(unsigned char) * input.rows * input.cols;
		hipMemcpy( (unsigned char*)&mVec[0], d_img, size, hipMemcpyDeviceToHost);
		size = sizeof(int) * input.rows * input.cols;
		hipMemcpy(outClusters, clusterIDs, size, hipMemcpyDeviceToHost);
		hipFree(clustMean); hipFree(clusterIDs); hipFree(clusterSize); hipFree(newIDs); hipFree(d_img);
		Mat retImg = Mat(input.rows, input.cols, CV_8UC1);
		for (int r = 0; r < input.rows; r++) {
			for (int c = 0; c < input.cols; c++) {
				int index = input.cols * r + c;
				 retImg.at<uchar>(r, c) = mVec[index];
			}
		}

		return std::make_pair(retImg, outClusters);
	}

	std::vector<float> meanTest(cv::Mat input) {
		input = Mat(4, 4, CV_8UC1);
		//std::vector<uchar> mVec((input.rows * input.cols), 0);
		std::vector<uchar> mVec(16, 1);
		mVec[5] = 5; mVec[6] = 5; mVec[9] = 5; mVec[10] = 5;
		for (int r = 0; r < input.rows; r++) {
			for (int c = 0; c < input.cols; c++) {
				int index = input.cols * r + c;
				input.at<uchar>(r, c) = mVec[index];
			}
		}
		//if (inputBW.isContinuous()) {
			//mVec.assign(inputBW.data, inputBW.data + inputBW.total());
		//}
		//else {
		for (int r = 0; r < input.rows; r++) {
			for (int c = 0; c < input.cols; c++) {
				int index = input.cols * r + c;
				mVec[index] = input.at<uchar>(r, c);
			}
		}
		//}

		unsigned char* d_img;
		float* clustMean;
		int* clusterSize, * clusterIDs, * newIDs;
		int size = sizeof(unsigned char) * input.rows * input.cols;
		hipMalloc((void**)&d_img, size);
		hipMemcpy(d_img, (unsigned char*)&mVec[0], size, hipMemcpyHostToDevice);
		size = sizeof(int) * input.rows * input.cols;
		hipMalloc((void**)&clusterSize, size); hipMalloc((void**)&clusterIDs, size); hipMalloc((void**)&newIDs, size);
		size = sizeof(float) * input.rows * input.cols;
		hipMalloc((void**)&clustMean, size);
		std::pair<float, float> imStat = varNOMEM(d_img, input.rows, input.cols);
		//float diff = imStat.first * 0.5;
		float diff = 6;
		printf("var: %f,  mean: %f\n", imStat.first, imStat.second);
		int numBlocksH = input.cols / BLOCK_SIZE;
		if (input.cols % BLOCK_SIZE > 0) {
			numBlocksH++;
		}
		int numBlocksV = input.rows / BLOCK_SIZE;
		if (input.rows % BLOCK_SIZE > 0) {
			numBlocksV++;
		}
		dim3 gGrid(numBlocksV, numBlocksH);
		dim3 bGrid(BLOCK_SIZE, BLOCK_SIZE);

		clusterMeanInit << <gGrid, bGrid >> > (input.rows, input.cols, clustMean, clusterSize, clusterIDs, newIDs, d_img);
		ngbrCompLE << < gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, diff, newIDs, clustMean);
		ngbrCompRE << < gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, diff, newIDs, clustMean);

		

		clusterMeanSum << <gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, clustMean, newIDs);
		clusterMeanDiv << <gGrid, bGrid >> > (clustMean, input.rows, input.cols, clusterSize);
		clusterSync << <gGrid, bGrid >> > (clusterIDs, input.rows, input.cols, newIDs, clusterSize, clustMean);
		ngbrCompUE << < gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, diff, newIDs, clustMean);
		ngbrCompDE << < gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, diff, newIDs, clustMean);

		

		clusterMeanSum << <gGrid, bGrid >> > (d_img, clusterIDs, clusterSize, input.rows, input.cols, clustMean, newIDs);
		clusterMeanDiv << <gGrid, bGrid >> > (clustMean, input.rows, input.cols, clusterSize);
		clusterSync << <gGrid, bGrid >> > (clusterIDs, input.rows, input.cols, newIDs, clusterSize, clustMean);
		imOut << <gGrid, bGrid >> > (d_img, clusterIDs, clustMean, input.rows, input.cols);

		size = sizeof(unsigned char) * input.rows * input.cols;
		hipMemcpy((unsigned char*)&mVec[0], d_img, size, hipMemcpyDeviceToHost);

		hipFree(clustMean); hipFree(clusterIDs); hipFree(clusterSize); hipFree(newIDs); hipFree(d_img);

		for (int i = 0; i < input.rows; i++) {
			for (int j = 0; j < input.cols; j++) {
				int index = input.cols * i + j;
				printf("%d, ", mVec[index]);
			}
			printf("\n");
		}
		return Mat();
	}
}